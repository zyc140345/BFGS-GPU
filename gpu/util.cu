//
// Created by 张易诚 on 24-10-12.
//

#include "util.h"

void RecordStartTime(hipEvent_t start, hipStream_t s) {
    CUDA_CHECK(hipEventRecord(start, s));
}

float RecordStopTime(hipEvent_t start, hipEvent_t stop, hipStream_t s) {
    CUDA_CHECK(hipEventRecord(stop, s));
    CUDA_CHECK(hipEventSynchronize(stop));
    float elapsedTime;
    CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    return elapsedTime / 1000;
}

static const char *cublasGetErrorString(hipblasStatus_t error) {
    switch (error) {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

void CublasCheck(hipblasStatus_t err, const char *file, int line) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        printf("%s in %s at line %d\n", cublasGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

void CudaCheck(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}